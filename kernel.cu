#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "kernel.h"

__global__ void mandelbrot_kernel(int *canvas, int *num_it, double l_margin, double r_margin, double u_margin, double d_margin, int N, double log_log_bailout, double ln2, double ln2_inv)
{
	int num_rows = blockDim.y*gridDim.y;
	int num_cols = blockDim.x*gridDim.x;
	double z_n_x = 0;
	double z_n_y = 0;
	double tmp_x, tmp_y;
	int tid_x = blockIdx.x*blockDim.x + threadIdx.x;
	int tid_y = blockIdx.y*blockDim.y + threadIdx.y;
	double c_x = l_margin + (tid_x/(double)(num_cols -1))*(r_margin - l_margin);
	double c_y = d_margin + (tid_y/(double)(num_rows -1))*(u_margin - d_margin);
	int escape_time = 0;
	int idx = tid_y*num_cols + tid_x;

	while(z_n_x*z_n_x + z_n_y*z_n_y < bailout && escape_time<N)
	{
		tmp_x = z_n_x*z_n_x - z_n_y*z_n_y;
		tmp_y = 2*z_n_x*z_n_y;
		z_n_x = tmp_x + c_x;
		z_n_y = tmp_y + c_y;
		escape_time ++;
	}
	if (escape_time>=N)
		canvas[idx] = -1;
	else
	{
		// double mod = sqrt(z_n_x*z_n_x + z_n_y*z_n_y);
		// double c = (log_log_bailout - log(log(mod))) * ln2_inv;
		// double ff = (log(c/64+1)/ln2+0.45);
		// double fc = (c*0.8 + escape_time)*100;
		// ff = ff - trunc(ff);

		// fc = ((int)fc)%GRADIENTLENGTH;
		// int color_idx = ff*GRADIENTLENGTH + 0.5;
		// int color_idx = ((int)fc);
		canvas[idx] = escape_time;
		// canvas[idx] = (int)(((escape_time - log(log(mod))/log(2.0))/(double)N)*255.0);
		// canvas[idx] = (int)(((double)escape_time/(double) N)*255.0);
	}
	num_it[idx] = escape_time;
}

void render(int *h_canvas, double zoom, long double center_x,long double center_y, double init_len, int dim_x, int dim_y, double N)
{
	double ln2_inv = 1.44269504088896340735992468100189213742664595415299;
	double ln2 = 0.69314718055994530941723212145817656807550013436026;
	hipError_t err = hipSuccess;
	double l_margin = center_x - init_len/2.0;
	double r_margin = center_x + init_len/2.0;
	double u_margin = center_y + init_len/2.0;
	double d_margin = center_y - init_len/2.0;
	
	
	dim3 threads_per_block(32,32,1);
	dim3 blocks_per_grid(dim_x/32,dim_y/32,1);
	
	size_t canvas_size =  dim_x*dim_y*sizeof(int);
	// if(dim_x%16!=0 || dim_y%16!=0)
	// {	

	// 	canvas_size = (16-(dim_x%16) + dim_x)*(16-(dim_y%16) + dim_y)*sizeof(int); 
	// }
	

	int *h_num_it = (int*)malloc(canvas_size);
	memset(h_num_it, 0, canvas_size);

	int *d_canvas = NULL;
	err = hipMalloc((void **)&d_canvas, canvas_size);
	if(err != hipSuccess)
	{
		printf("Error in hipMalloc : d_canvas\n");
		exit(EXIT_FAILURE);
	}

	int *d_num_it = NULL;
	err = hipMalloc((void **)&d_num_it, canvas_size);
	if(err != hipSuccess)
	{
		printf("Error in hipMalloc : d_num_it\n");
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_canvas, h_canvas, canvas_size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("Error in hipMemcpy : d_canvas\n");
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_num_it, h_num_it, canvas_size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("Error in hipMemcpy : d_num_it\n");
		exit(EXIT_FAILURE);
	}
	double log_log_bailout = log(log(bailout));
	mandelbrot_kernel <<<blocks_per_grid, threads_per_block>>> (d_canvas, d_num_it, l_margin, r_margin, u_margin, d_margin, N, log_log_bailout, ln2, ln2_inv);

	err = hipGetLastError();
	if(err!=hipSuccess)
	{
		printf("Error in kernel\n");
		exit(EXIT_FAILURE);
	}

	// printf("Getting the canvas back from kernel\n");
	// fflush(stdout);

	err = hipMemcpy(h_canvas, d_canvas, canvas_size, hipMemcpyDeviceToHost);
	if(err!=hipSuccess)
	{
		printf("Error in hipMemcpy: h_canvas\n");
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(h_num_it, d_num_it, canvas_size, hipMemcpyDeviceToHost);
	if(err!=hipSuccess)
	{
		printf("Error in hipMemcpy: h_num_it\n");
		exit(EXIT_FAILURE);
	}

	// printf("Freeing device memory\n");
	// fflush(stdout);
	err = hipFree(d_canvas);
	if(err!=hipSuccess)
	{
		printf("Error in hipFree: d_canvas\n");
		exit(EXIT_FAILURE);
	}

	err = hipDeviceReset();
	if(err!=hipSuccess)
	{
		printf("Error in hipDeviceReset\n");
		exit(EXIT_FAILURE);
	}	

	// printf("analyzing escape times\n");
	// fflush(stdout);
	
	// int max_esc = 0;
	// int min_esc = 1000;
	// double avg_esc = 0.0;
	// int outside_count = 0;
	// for(int i=0;i<dim_y;i++)
	// {
	// 	for(int j=0;j<dim_x;j++)
	// 	{
	// 		if (h_canvas[i*dim_x + j]!=0)
	// 		{
	// 			if (h_num_it[i*dim_x + j]>max_esc)
	// 				max_esc = h_num_it[i*dim_x + j];
	// 			if (h_num_it[i*dim_x + j]<min_esc)
	// 				min_esc = h_num_it[i*dim_x + j];
	// 			avg_esc += h_num_it[i*dim_x + j];
	// 			outside_count ++;
	// 		}
	// 	}
	// }
	// avg_esc = avg_esc/outside_count;
	// printf("max it:%d, min_it:%d, outside_count:%d, avg_it:%lf",max_esc, min_esc, outside_count, avg_esc);


	
}